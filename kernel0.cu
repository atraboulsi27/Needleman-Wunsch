#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"

#define BLOCK_SIZE 1024

__global__ void nw_kernel(unsigned char* reference, unsigned char* query, int* matrix, unsigned int N, int iteration_number) {


    for( int diagonal = 0; diagonal < BLOCK_SIZE; diagonal++ ) {

        // Verify that the diagonal thread index does not exceed the maximum number of elements allowed by the diagonal at this iteration.
        if( threadIdx.x <= diagonal  ) {

            // Get the position of the thread inside the block.
            int pos_in_block_x = threadIdx.x;
            int pos_in_block_y = diagonal - pos_in_block_x;

            // Calculate the positions of the thread inside the matrix.
            int mat_row = iteration_number * blockDim.x + pos_in_block_y;
            int mat_col = iteration_number * blockDim.x + pos_in_block_x;
            
            if( mat_row < N && mat_col < N ) {

                // Calculate value left, top, and top-left neighbors.
                int top = 
                    (mat_row == 0) ? 
                        ((mat_col + 1)*DELETION) : matrix[ (mat_row - 1)*N + mat_col ];
                
                int left = 
                    (mat_col == 0) ? 
                        ((mat_row + 1)*INSERTION) : matrix[ mat_row*N + (mat_col - 1) ];
                
                int topleft = 
                    (mat_row == 0) ? 
                        (mat_col*DELETION) : (mat_col == 0) ? 
                            (mat_row*INSERTION) : matrix[ (mat_row - 1)*N + (mat_col - 1) ];

                // Determine scores of the three possible outcomes: insertion, deletion, and match.
                int insertion = top  + INSERTION;
                int deletion  = left + DELETION;

                // Get the characters to verify if there is a match.
                char ref_char   = reference[mat_col];
                char query_char = query[mat_row];

                int match = topleft + ( (ref_char == query_char) ? MATCH : MISMATCH );
                
                // Select the maximum between the three.
                int max = (insertion > deletion) ? insertion : deletion;
                max = (match > max) ? match : max; 

                // Update the matrix at the correct position
                matrix[  mat_row*N + mat_col ] = max;
                
            }
        }

        __syncthreads();

    }

    for( int diagonal = BLOCK_SIZE; diagonal < 2*BLOCK_SIZE; diagonal++ ) {

        // Verify that the diagonal thread index does not exceed the maximum number of elements allowed by the diagonal at this iteration.
        if( threadIdx.x < 2*BLOCK_SIZE - diagonal ) {

            // Get the position of the thread inside the block.
            int pos_in_block_x = BLOCK_SIZE - threadIdx.x - 1;
            int pos_in_block_y = diagonal - pos_in_block_x - 1;

            // Calculate the positions of the thread inside the matrix.
            int mat_row = iteration_number * blockDim.x + pos_in_block_y;
            int mat_col = iteration_number * blockDim.x + pos_in_block_x;

            if( mat_row < N && mat_col < N ) {

                // Calculate value left, top, and top-left neighbors.
                int top = 
                    (mat_row == 0) ? 
                        ((mat_col + 1)*DELETION) : matrix[ (mat_row - 1)*N + mat_col ];
                
                int left = 
                    (mat_col == 0) ? 
                        ((mat_row + 1)*INSERTION) : matrix[ mat_row*N + (mat_col - 1) ];
                
                int topleft = 
                    (mat_row == 0) ? 
                        (mat_col*DELETION) : (mat_col == 0) ? 
                            (mat_row*INSERTION) : matrix[ (mat_row - 1)*N + (mat_col - 1) ];

                // Determine scores of the three possible outcomes: insertion, deletion, and match.
                int insertion = top  + INSERTION;
                int deletion  = left + DELETION;

                // Get the characters to verify if there is a match.
                char ref_char   = reference[mat_col];
                char query_char = query[mat_row];

                int match = topleft + ( (ref_char == query_char) ? MATCH : MISMATCH );
                
                // Select the maximum between the three.
                int max = (insertion > deletion) ? insertion : deletion;
                max = (match > max) ? match : max; 

                if (mat_row == 0 && mat_col == 1023)
                    printf("row: %d, col: %d, max: %d, del: %d, ins: %d, match: %d, %c, %c\n", mat_row, mat_col, max, deletion, insertion, match, ref_char, query_char);

                // Update the matrix at the correct position
                matrix[  mat_row*N + mat_col ] = max;
                
            }
        }

        __syncthreads();
    }
}


void nw_gpu0(unsigned char* reference_d, unsigned char* query_d, int* matrix_d, unsigned int N) { 
    
    unsigned int numThreadsPerBlock = BLOCK_SIZE;

    for(int iter=0; iter < 2*(N + BLOCK_SIZE - 1) / BLOCK_SIZE; iter++) {

        // Configure next run
        unsigned int numBlocks = (iter < (N + BLOCK_SIZE - 1) / BLOCK_SIZE) ? (iter + 1) : (2 * (N + BLOCK_SIZE - 1) / BLOCK_SIZE - iter - 1);
      
        printf("%d\n", numBlocks);
        // Launch kernel
        nw_kernel<<<numBlocks, numThreadsPerBlock>>>(reference_d, query_d, matrix_d, N, iter);
        
        hipDeviceSynchronize();

    }

}

